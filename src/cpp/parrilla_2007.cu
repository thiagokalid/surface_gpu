#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>

#define BLOCK_SIZE 32

extern "C" {

    // Device function to calculate the value of tof
    __device__ double tof(int k, float* xS, float* zS, float x1, float z1, float c) {
        // Calculate Mk
        double Mk = (zS[k + 1] - zS[k]) / (xS[k + 1] - xS[k]);

        // Return the result
        return (1 / c) * ((xS[k] - x1) + Mk * (zS[k] - z1)) / sqrt(pow(xS[k] - x1, 2) + pow(zS[k] - z1, 2));
    }

    // Device function to compute the step
    __device__ int step(int k0, float xA, float zA, float xF, float zF, float* xS, float* zS, float c1, float c2) {
        // Compute Vk0 and Vk using the tof function
        double Vk0 = tof(k0, xS, zS, xA, zA, c1) + tof(k0, xS, zS, xF, zF, c2);
        double Vk = tof(k0 + 1, xS, zS, xA, zA, c1) + tof(k0 + 1, xS, zS, xF, zF, c2);

        // The step difference is simply the difference between Vk and Vk0
        return round(Vk0 / (Vk - Vk0));
    }

    // Global kernel function
    __global__ void _parrilla_2007(int* d_k, float* d_xA, float* d_zA, float* d_xF, float* d_zF, float* xS, float* zS, float c1, float c2, int Na, int Nf, int N, int maxIter, int epsilon) {
        // Compute position that this thread is responsible for
        int c = blockIdx.y * blockDim.y + threadIdx.y;
        int r = blockIdx.x * blockDim.x + threadIdx.x;

        // Solver parameters:
        bool converged = false;
        int k0 = 0;
        int k;


        // Newton-Raphson method:
        if( r < Na && c < Nf){
            double xA = d_xA[r];
            double zA = d_zA[r];
            double xF = d_xF[c];
            double zF = d_zF[c];

            for (int i = 0; i < maxIter; i++) {
                // Newton-step:
                // int istep = step(k0, xA, zA, xF, zF, xS, zS, c1, c2)
                int istep = step(k0, xA, zA, xF, zF, xS, zS, c1, c2);

                k = k0 - istep;

                // Check if k is within indexable bounds:
                if (k < 0) {
                    k = 0;
                } else if (k >= N-2) {
                    k = N - 3;
                }

                //printf("k = %d, k0 = %d; step = %d; abs(k - k0) = %d\n", k, k0, istep, abs(k - k0));

                // Check stopping criteria
                if (abs(k - k0) <= epsilon) {
                    converged = true;
                    break;
                }
                k0 = k;
            }
            int idx = c + r * Nf;
            //printf("%d\n", idx);
            d_k[idx] = k;
        }
    }

    // Host function to call the kernel
    int* parrilla_2007(float* xA, float* zA, float* xF, float* zF, float* xS, float* zS, float c1, float c2, int Na, int Nf, int N, int maxIter, int epsilon) {
        float *d_xS, *d_zS, *d_xF, *d_zF, *d_xA, *d_zA;

        int *d_k;
        int *k = new int[Na * Nf];

        // Surface: Allocate memory on the device
        hipMalloc(&d_xS, N * sizeof(float));
        hipMalloc(&d_zS, N * sizeof(float));
        hipMemcpy(d_xS, xS, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_zS, zS, N * sizeof(float), hipMemcpyHostToDevice);

        hipMalloc(&d_xA, Na * sizeof(float));
        hipMalloc(&d_zA, Na * sizeof(float));
        hipMemcpy(d_xA, xA, Na * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_zA, zA, Na * sizeof(float), hipMemcpyHostToDevice);

        hipMalloc(&d_xF, Nf * sizeof(float));
        hipMalloc(&d_zF, Nf * sizeof(float));
        hipMemcpy(d_xF, xF, Nf * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_zF, zF, Nf * sizeof(float), hipMemcpyHostToDevice);

        hipMalloc(&d_k, Na * Nf * sizeof(int));

        // Run the kernel:
        // Create as many blocks as necessary to map all of C
        int X = ceilf(Na/(float)BLOCK_SIZE);
        int Y = ceilf(Nf/(float)BLOCK_SIZE);
        printf("X = %d \n", X);
        printf("Y = %d \n", Y);

        dim3 gridDim(X, Y, 1);
        dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);

        // Call the kernel
        _parrilla_2007<<<gridDim, blockDim>>>(d_k, d_xA, d_zA, d_xF, d_zF, d_xS, d_zS, c1, c2, Na, Nf, N, maxIter, epsilon);

        // Check for any kernel errors
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        }

        // Wait for the kernel to finish
        hipDeviceSynchronize();

        // Copy the results back to host
        hipMemcpy(k, d_k, Na * Nf * sizeof(int), hipMemcpyDeviceToHost);


        // Free up memory allocated on GPU:
        hipFree(d_xS);
        hipFree(d_zS);
        hipFree(d_zA);
        hipFree(d_zA);
        hipFree(d_zF);
        hipFree(d_zF);
        hipFree(d_k);

        return k;
    }
}
